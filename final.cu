
#include <hip/hip_runtime.h>
#include <algorithm> 
#include <iostream> 
#include <vector>
using namespace std; 
  
typedef std::vector<double> vi; 
  
typedef vector<vector<double> > matrix; 
vi A; 
vi IA = { 0 }; 
vi JA; 
vi DA;
int length;

__global__ void multi(double *a, double *b, double *c, int n){
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if(id<n){
        c[id] = a[id]*b[id];
    }
    }
    

void printMatrix(const matrix& M) 
{ 
    int m = M.size();
    int n = M[0].size(); 
    for (int i = 0; i < m; i++) { 
        for (int j = 0; j < n; j++)  
            cout << M[i][j] << " ";         
        cout << endl; 
    }

} 
void printlist(double* V, char* msg, int m) 
{   
    cout << msg << "[ "; 
    
    for( int i =0; i<m; i++){ 
        cout << V[i] << " "; 
    }; 
    cout << "]" << endl; 
} 
void copyto(vi& V, double* C, int N){
    // cout<<"yo"<<std::endl;
    //
     //cout<<N<<std::endl;+-
    for(int i =0; i<=N-1; i++)
    std::copy(V.begin(), V.end(), C);
    //}
    //free(C);
}
void printVector(const vi& V, char* msg) 
{ 
  
    cout << msg << "[ "; 
    for_each(V.begin(), V.end(), [](double a) { 
        cout << a << " "; 
    }); 
    cout << "]" << endl; 
} 
void decirculate( vi& JA, const matrix& M, const vi& IA){

    int rows = M[0].size();
    int itr = IA.size();
    rows = rows-1;
    int i,j;
    for(i=0; i<itr-1; i++){
        for(j=IA[i];j<IA[i+1];j++){
            if(JA[j]<i){
                JA[j] = rows - (i-JA[j]-1);
            }
            else{
                JA[j] = JA[j]-i;

            }
        }
    }
    //return JA;
}
void extendVec(vi& A,int num,int size){
for(int i = 0; i<=num; i++){
cout<<num<<std::endl;
    for (int j =0; j<size; j++ ){
    A.push_back(A[j]);
    }
}
    
}
void createdense(matrix& CA, vi& JA){
    int m = JA.size();
    int n = CA[0].size();
    for(int i =0; i<n; i++){
        for(int k =0; k<m; k++){
        DA.push_back(CA[JA[k]][i]); } 
    }
    }


void sparesify(const matrix& M) 
{ 
    int m = M.size(); 
    int n = M[0].size(), i, j; 
    
    int dab = 0; 
  
    for (i = 0; i < m; i++) { 
        for (j = 0; j < n; j++) { 
            if (M[i][j] != 0) { 
                A.push_back(M[i][j]); 
                JA.push_back(j); 
  
               
                dab++; 
            } 
        }
       
        IA.push_back(dab); 
    } 
    decirculate(JA,M,IA); 
    printMatrix(M);
    cout<<"++++++++++++++++++++++++++++++++++++++++++"<<std::endl; 
    printVector(A, (char*)"A = "); 
    printVector(IA, (char*)"IA = "); 
    printVector(JA, (char*)"JA = ");
    cout<<"++++++++++++++++++++++++++++++++++++++++++"<<std::endl; 
} 
  

int main() 

{
    double *IN,*in; 
    double *OUT,*out;
    double *ANS,*ans;
     
     matrix M = { 
        { 0, 0, 0, 0, 1 }, 
        { 5, 8, 0, 0, 0 }, 
        { 0, 0, 3, 0, 0 }, 
        { 0, 6, 0, 0, 1 }, 
        };
        matrix CA = {{1,1,1,1,1},
    {2,2,2,2,2},
    {3,3,3,3,3},
    {4,4,4,4,4},
    {5,5,5,5,5}};
    sparesify(M);
        createdense(CA,JA); 
    extendVec(A,DA.size()/A.size(),A.size());
    cout<<DA.size()<<std::endl;
    length = DA.size();
    int size = length*sizeof(double);
        cout<<size<<std::endl;
    int gridsize;
    hipMalloc((void **) &in, size);
    hipMalloc((void **) &out, size);
    hipMalloc((void **) &ans, size);
    IN = (double *)malloc(size);
    OUT = (double *)malloc(size);
    ANS = (double *)malloc(size);
    hipMalloc((void **) &in, size);
    hipMalloc((void **) &out, size);
    hipMalloc((void **) &ans, size);

        printVector(DA, (char*)"DA = ");
    copyto(DA,&IN[0],DA.size());
    copyto(A,&OUT[0],DA.size());
    printlist(&OUT[0], (char*)"Out = ",DA.size());
    printlist(&IN[0], (char*)"IN = ",DA.size()); 
    cout<<"++++++++++++++++++++++++++++++++++++++++++"<<std::endl;
    gridsize =ceil(size/1024);
    gridsize = 32;
    hipMemcpy(in, IN, size, hipMemcpyHostToDevice);
    hipMemcpy(out,OUT, size, hipMemcpyHostToDevice);
    multi<<<32,1024>>>(in,out,ans,DA.size());
    hipMemcpy(ANS, ans, size, hipMemcpyDeviceToHost);
    printlist(&ANS[0], (char*)"ANS = ",DA.size()); 
    free (IN); free(OUT); free(ANS);
    hipFree(in); hipFree(out); hipFree(ans);
    return 0; 
} 
